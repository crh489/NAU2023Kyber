
// @Author: Arpan Jati
// Adapted from NewHope Reference Codebase and Parallelized using CUDA
// Updated : August 2019
// -------------------------------------------------------------
// CODE FOR PERFORMANCE COMPARISON. NOT FOR ACTUAL DEPLOYMENT
// -------------------------------------------------------------

#include "main.h";
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include "rng.h"
#include "api.h"
#include "params.h"
#include "indcpa.h"
#include <chrono>
#include <iostream>
#define	MAX_MARKER_LEN		50
#define KAT_SUCCESS          0
#define KAT_FILE_OPEN_ERROR -1
#define KAT_DATA_ERROR      -3
#define KAT_CRYPTO_FAILURE  -4

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;
// MAIN CPA 

void allocatePolySet(poly_set4* polySet)
{
	HANDLE_ERROR(hipMalloc(&(polySet->a), sizeof(poly)));
	HANDLE_ERROR(hipMalloc(&(polySet->b), sizeof(poly)));
	HANDLE_ERROR(hipMalloc(&(polySet->c), sizeof(poly)));
	HANDLE_ERROR(hipMalloc(&(polySet->d), sizeof(poly)));

	HANDLE_ERROR(hipMalloc(&(polySet->av), sizeof(polyvec)));
	HANDLE_ERROR(hipMalloc(&(polySet->bv), sizeof(polyvec)));
	HANDLE_ERROR(hipMalloc(&(polySet->cv), sizeof(polyvec)));
	HANDLE_ERROR(hipMalloc(&(polySet->dv), sizeof(polyvec)));
	HANDLE_ERROR(hipMalloc(&(polySet->ev), sizeof(polyvec)));
	HANDLE_ERROR(hipMalloc(&(polySet->fv), sizeof(polyvec)));

	HANDLE_ERROR(hipMalloc(&(polySet->AV), sizeof(polyvec) * 4));

	HANDLE_ERROR(hipMalloc(&(polySet->seed), (KYBER_SYMBYTES * 2) * N_TESTS));

	HANDLE_ERROR(hipMalloc(&(polySet->large_buffer_a), LARGE_BUFFER_SZ * N_TESTS));
	HANDLE_ERROR(hipMalloc(&(polySet->large_buffer_b), LARGE_BUFFER_SZ * N_TESTS));
}

void freePolySet(poly_set4* polySet)
{
	HANDLE_ERROR(hipFree(polySet->a));
	HANDLE_ERROR(hipFree(polySet->b));
	HANDLE_ERROR(hipFree(polySet->c));
	HANDLE_ERROR(hipFree(polySet->d));

	HANDLE_ERROR(hipFree(polySet->av));
	HANDLE_ERROR(hipFree(polySet->bv));
	HANDLE_ERROR(hipFree(polySet->cv));
	HANDLE_ERROR(hipFree(polySet->dv));
	HANDLE_ERROR(hipFree(polySet->ev));
	HANDLE_ERROR(hipFree(polySet->fv));

	HANDLE_ERROR(hipFree(polySet->seed));

	HANDLE_ERROR(hipFree(polySet->large_buffer_a));
	HANDLE_ERROR(hipFree(polySet->large_buffer_b));
}

int COUNT = 1;

int kybercuda(int kyber_implementation, char* input_file_path, char* encryption_file_path, char* decryption_file_path, int mode, int numThreads)
{
	FILE* f_input;
	FILE* f_encryption;
	FILE* f_decryption;
	uint8_t header[54] = "";
	size_t cipherfile_size;
	uint8_t* cipherfile;
	uint8_t* cipherfile_d;
	long long fileSize, originalFileSize;
	unsigned char* pk_h_0;
	unsigned char* sk_h_0;
	unsigned char* coins_h_0;
	unsigned char* rng_buf_h_0;
	unsigned char* pk_d_0;
	unsigned char* sk_d_0;
	unsigned char* coins_d_0;
	unsigned char* rng_buf_d_0;
	uint8_t* image_d;
	uint8_t* dec_image_d;
	hipEvent_t start, stop, start1, stop1, start2, stop2, start3, stop3;
	uint8_t* image;
	uint8_t* dec_image;
	/*

	Read Input File
	*/
	f_input = fopen(input_file_path, "rb");
	f_encryption = fopen(encryption_file_path, "wb+");
	f_decryption = fopen(decryption_file_path, "wb+");
	if (!f_input) {
		printf("Invalid file path, file might not exist: %s\nProgram End.\n", input_file_path);
		return 0;
	}
	_fseeki64(f_input, 0, SEEK_END);
	fileSize = _ftelli64(f_input);
	originalFileSize = fileSize;
	_fseeki64(f_input, 0, SEEK_SET);
	image = (uint8_t*)calloc(1, fileSize);
	dec_image = (uint8_t*)calloc(1, fileSize*2);
	fread(header, 1, 54, f_input);
	fwrite(header, 54, 1, f_encryption);
	fwrite(header, 54, 1, f_decryption);
	_fseeki64(f_input, 0, SEEK_SET);
	fread(image, 1, fileSize, f_input);
	_fseeki64(f_input, 0, SEEK_SET);
	cipherfile_size = fileSize * (KYBER_INDCPA_BYTES / KYBER_INDCPA_MSGBYTES);
	
	HANDLE_ERROR(hipSetDevice(0));
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventCreate(&start3);
	hipEventCreate(&stop3);
	HANDLE_ERROR(hipHostAlloc((void**)&pk_h_0, KYBER_INDCPA_PUBLICKEYBYTES, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&sk_h_0, KYBER_INDCPA_SECRETKEYBYTES, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&coins_h_0, KYBER_SYMBYTES, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&rng_buf_h_0, KYBER_SYMBYTES * 2, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&cipherfile, cipherfile_size, hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&pk_d_0, KYBER_INDCPA_PUBLICKEYBYTES));
	HANDLE_ERROR(hipMalloc((void**)&sk_d_0, KYBER_INDCPA_SECRETKEYBYTES));
	HANDLE_ERROR(hipMalloc((void**)&coins_d_0, KYBER_SYMBYTES));
	HANDLE_ERROR(hipMalloc((void**)&rng_buf_d_0, KYBER_SYMBYTES * 2));
	HANDLE_ERROR(hipMalloc((void**)&image_d, fileSize));
	HANDLE_ERROR(hipMalloc((void**)&dec_image_d, fileSize*2));
	HANDLE_ERROR(hipMalloc((void**)&cipherfile_d, cipherfile_size));
	memset(coins_h_0, 0, KYBER_SYMBYTES);
	randombytes(rng_buf_h_0, KYBER_SYMBYTES * 2);
	hipStream_t stream_0;
	HANDLE_ERROR(hipStreamCreate(&stream_0));
	poly_set4 tempPoly_0;
	hipEventRecord(start);
	HANDLE_ERROR(hipMemcpyAsync(pk_d_0, pk_h_0, KYBER_INDCPA_PUBLICKEYBYTES, hipMemcpyHostToDevice, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(sk_d_0, sk_h_0, KYBER_INDCPA_SECRETKEYBYTES, hipMemcpyHostToDevice, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(coins_d_0, coins_h_0, KYBER_SYMBYTES, hipMemcpyHostToDevice, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(rng_buf_d_0, rng_buf_h_0, KYBER_SYMBYTES * 2, hipMemcpyHostToDevice, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(image_d, image, fileSize, hipMemcpyHostToDevice, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(dec_image_d, dec_image, fileSize*2, hipMemcpyHostToDevice, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(cipherfile_d, cipherfile, cipherfile_size, hipMemcpyHostToDevice, stream_0));
	size_t fileSize_d = fileSize;
	size_t cipherfile_size_d = cipherfile_size;
	allocatePolySet(&tempPoly_0);
	indcpa_keypair(1, &tempPoly_0, pk_d_0, sk_d_0, rng_buf_d_0, stream_0);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	/*
	
	ENCRYPT
	
	*/
	hipEventRecord(start1);
	for (long long i = 0; i < fileSize_d - 54; i += KYBER_INDCPA_MSGBYTES) {
		indcpa_enc(1, &tempPoly_0, &cipherfile_d[i * (KYBER_INDCPA_BYTES / KYBER_INDCPA_MSGBYTES)], &image_d[i+54], pk_d_0, coins_d_0, stream_0);
	}
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	float ms = 0;
	hipEventElapsedTime(&ms, start1, stop1);
	printf("\nEncryption Finished, %f\n\n", ms);
	/*
	
	DECRYPT
	
	*/
	hipEventRecord(start2);
	for (long long i = 0; i < cipherfile_size_d; i += KYBER_INDCPA_BYTES) {
		indcpa_dec(1, &tempPoly_0, &dec_image_d[i / (KYBER_INDCPA_BYTES / KYBER_INDCPA_MSGBYTES)], &cipherfile_d[i], sk_d_0, stream_0);
	}
	hipEventRecord(stop2);
	hipEventSynchronize(stop2);
	ms = 0;
	hipEventElapsedTime(&ms, start2, stop2);
	printf("\nDecryption Finished, %f (ms)\n\n", ms);
	hipEventRecord(start3);
	HANDLE_ERROR(hipMemcpyAsync(pk_h_0, pk_d_0, KYBER_INDCPA_PUBLICKEYBYTES, hipMemcpyDeviceToHost, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(sk_h_0, sk_d_0, KYBER_INDCPA_SECRETKEYBYTES, hipMemcpyDeviceToHost, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(coins_h_0, coins_d_0, KYBER_SYMBYTES, hipMemcpyDeviceToHost, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(rng_buf_h_0, rng_buf_d_0, KYBER_SYMBYTES * 2, hipMemcpyDeviceToHost, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(image, image_d, fileSize, hipMemcpyDeviceToHost, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(dec_image, dec_image_d, fileSize*2, hipMemcpyDeviceToHost, stream_0));
	HANDLE_ERROR(hipMemcpyAsync(cipherfile, cipherfile_d, cipherfile_size, hipMemcpyDeviceToHost, stream_0));
	hipEventRecord(stop3);
	hipEventSynchronize(stop3);
	HANDLE_ERROR(hipStreamDestroy(stream_0));
	fwrite(cipherfile, cipherfile_size, 1, f_encryption);
	fclose(f_encryption);
	fclose(f_input);
	fwrite(dec_image, fileSize, 1, f_decryption);
	fclose(f_decryption);
	HANDLE_ERROR(hipFree(pk_d_0));
	HANDLE_ERROR(hipFree(sk_d_0));
	HANDLE_ERROR(hipFree(coins_d_0));
	HANDLE_ERROR(hipFree(rng_buf_d_0));
	HANDLE_ERROR(hipFree(image_d));
	HANDLE_ERROR(hipFree(dec_image_d));
	HANDLE_ERROR(hipFree(cipherfile_d));
	HANDLE_ERROR(hipHostFree(pk_h_0));
	HANDLE_ERROR(hipHostFree(sk_h_0));
	HANDLE_ERROR(hipHostFree(coins_h_0));
	HANDLE_ERROR(hipHostFree(rng_buf_h_0));
	HANDLE_ERROR(hipHostFree(cipherfile));
	memset(image, 0, sizeof(*image));
	free(image);
	freePolySet(&tempPoly_0);
	HANDLE_ERROR(hipDeviceSynchronize());
	return 0;
}